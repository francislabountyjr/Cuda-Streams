#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <cstdio>

using namespace std;

__global__ void test_kernel(int step)
{
	printf("Loop: %d\n", step);
}

int main()
{
	int n_stream = 5;
	hipStream_t* ls_stream;
	ls_stream = (hipStream_t*)new hipStream_t[n_stream];

	// Create multiple streams
	for (int i = 0; i < n_stream; i++)
	{
		hipStreamCreate(&ls_stream[i]);
	}

	// Execute kernel with each CUDA stream
	for (int i = 0; i < n_stream; i++)
	{
		if (i == 3)
		{
			test_kernel << <1, 1, 0, 0 >> > (i); // Default stream is synchronous so no other streams will be launched until completed
		}
		test_kernel << <1, 1, 0, ls_stream[i] >> > (i);
	}

	// Synchronize host and GPU
	hipDeviceSynchronize();

	// Cleanup
	for (int i = 0; i < n_stream; i++)
	{
		hipStreamDestroy(ls_stream[i]);
	}
	delete[] ls_stream;
}
